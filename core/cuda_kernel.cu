#include "hip/hip_runtime.h"
#include <assert.h>
#include "core.h"
#include "cuda_kernel.h"

std::vector<char*> local_buffer;
size_t local_buffer_size;
hipStream_t cuda_stream_array[8];

#define UNROLL_1  1
#define UNROLL_4  4
#define UNROLL_8  8
#define UNROLL_16 16

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}                                                               \

__global__ void execute_kernel_compute_cuda_kernel_unroll_1(long iter, double *A);
__global__ void execute_kernel_compute_cuda_kernel_unroll_4(long iter, double *A);
__global__ void execute_kernel_compute_cuda_kernel_unroll_8(long iter, double *A);
__global__ void execute_kernel_compute_cuda_kernel_unroll_16(long iter, double *A);

void init_cuda_support(const std::vector<TaskGraph> &graphs, const std::vector<int> &local_gpus)
{
  int nb_gpus = local_gpus.size();
  
  local_buffer.reserve(nb_gpus);
  int nb_blocks = graphs[0].kernel.nb_blocks;
  int threads_per_block = graphs[0].kernel.threads_per_block;
  int cuda_unroll = graphs[0].kernel.cuda_unroll;
  // printf("init cuda support nb_gpus %d nb_blocks %d, threads_per_block %d, cuda_unroll %d\n", nb_gpus, nb_blocks, threads_per_block, cuda_unroll);
  local_buffer_size = nb_blocks * threads_per_block * sizeof(double);
  for (int i = 0; i < nb_gpus; i++) {
    gpuErrchk( hipSetDevice(local_gpus[i]) );
    gpuErrchk( hipMalloc((void**)&(local_buffer[i]), sizeof(double) * nb_blocks * threads_per_block * cuda_unroll) );
    assert(local_buffer[i] != NULL);
    gpuErrchk( hipStreamCreate(&(cuda_stream_array[i])) );
  }
}

void fini_cuda_support(const std::vector<int> &local_gpus)
{
  // printf("fini cuda support nb_gpus %d\n", local_gpus.size());
  for (int i = 0; i < local_buffer.size(); i++) {
    gpuErrchk( hipSetDevice(local_gpus[i]) );
    gpuErrchk( hipFree(local_buffer[i]) );
    local_buffer[i] = NULL;
    gpuErrchk( hipStreamDestroy(cuda_stream_array[i]) );
  }
  local_buffer.clear();
}

void execute_kernel_compute_cuda(const Kernel &kernel, char *scratch_ptr, size_t scratch_bytes, int gpu_id)
{
  // printf("CUDA COMPUTE KERNEL buffer %p, size %lld, nb_blocks %d, threads_per_block %d\n", scratch_ptr, scratch_bytes, kernel.nb_blocks, kernel.threads_per_block);
  assert(scratch_bytes <= local_buffer_size);

  if (kernel.memcpy_required == 1) {
 //   printf("enable memcpy in\n");
    gpuErrchk( hipMemcpyAsync(local_buffer[gpu_id], scratch_ptr, scratch_bytes, hipMemcpyHostToDevice, cuda_stream_array[gpu_id]) ); 
    gpuErrchk( hipStreamSynchronize(cuda_stream_array[gpu_id]) );
  }
  if (kernel.cuda_unroll == 4) {
    execute_kernel_compute_cuda_kernel_unroll_4<<<kernel.nb_blocks, kernel.threads_per_block, 0, cuda_stream_array[gpu_id]>>>(kernel.iterations, (double *)local_buffer[gpu_id]);
  } else if (kernel.cuda_unroll == 8) {
    execute_kernel_compute_cuda_kernel_unroll_8<<<kernel.nb_blocks, kernel.threads_per_block, 0, cuda_stream_array[gpu_id]>>>(kernel.iterations, (double *)local_buffer[gpu_id]);
  } else if (kernel.cuda_unroll == 16) {
    execute_kernel_compute_cuda_kernel_unroll_16<<<kernel.nb_blocks, kernel.threads_per_block, 0, cuda_stream_array[gpu_id]>>>(kernel.iterations, (double *)local_buffer[gpu_id]);
  } else {
    execute_kernel_compute_cuda_kernel_unroll_1<<<kernel.nb_blocks, kernel.threads_per_block, 0, cuda_stream_array[gpu_id]>>>(kernel.iterations, (double *)local_buffer[gpu_id]);
  }
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipStreamSynchronize(cuda_stream_array[gpu_id]) );
  if (kernel.memcpy_required == 1) {
 //   printf("enable memcpy out\n");
    gpuErrchk( hipMemcpyAsync(scratch_ptr, local_buffer[gpu_id], scratch_bytes, hipMemcpyDeviceToHost, cuda_stream_array[gpu_id]) );
    gpuErrchk( hipStreamSynchronize(cuda_stream_array[gpu_id]) );
  }
}

void execute_kernel_compute_cuda(const Kernel &kernel, char *scratch_ptr, size_t scratch_bytes, char *device_ptr, size_t device_bytes) {
  printf("CUDA COMPUTE KERNEL scratch_ptr %p, size %lld, nb_blocks %d, threads_per_block %d, device_ptr %p, size %lld\n", scratch_ptr, scratch_bytes, kernel.nb_blocks, kernel.threads_per_block, device_ptr, device_bytes);
  assert(scratch_bytes <= device_bytes);

  if (kernel.memcpy_required == 1) {
 //   printf("enable memcpy in\n");
    gpuErrchk( hipMemcpyAsync(device_ptr, scratch_ptr, scratch_bytes, hipMemcpyHostToDevice, 0) ); 
    gpuErrchk( hipStreamSynchronize(0) );
  }
  if (kernel.cuda_unroll == 4) {
    execute_kernel_compute_cuda_kernel_unroll_4<<<kernel.nb_blocks, kernel.threads_per_block, 0, 0>>>(kernel.iterations, (double *)device_ptr);
  } else if (kernel.cuda_unroll == 8) {
    execute_kernel_compute_cuda_kernel_unroll_8<<<kernel.nb_blocks, kernel.threads_per_block, 0, 0>>>(kernel.iterations, (double *)device_ptr);
  } else if (kernel.cuda_unroll == 16) {
    execute_kernel_compute_cuda_kernel_unroll_16<<<kernel.nb_blocks, kernel.threads_per_block, 0, 0>>>(kernel.iterations, (double *)device_ptr);
  } else {
    execute_kernel_compute_cuda_kernel_unroll_1<<<kernel.nb_blocks, kernel.threads_per_block, 0, 0>>>(kernel.iterations, (double *)device_ptr);
  }
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipStreamSynchronize(0) );
  if (kernel.memcpy_required == 1) {
 //   printf("enable memcpy out\n");
    gpuErrchk( hipMemcpyAsync(scratch_ptr, device_ptr, scratch_bytes, hipMemcpyDeviceToHost, 0) );
    gpuErrchk( hipStreamSynchronize(0) );
  }
}

__global__ void execute_kernel_compute_cuda_kernel_unroll_1(long iter, double *A)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  double A_local = A[tid];
  for (int i = 0; i < iter; i++) {
    A_local = fma(A_local, A_local, A_local);
  }
  A[tid] = A_local;
}

__global__ void execute_kernel_compute_cuda_kernel_unroll_4(long iter, double *A)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  double A_local[UNROLL_4];
  int u;
  #pragma unroll
  for (u = 0; u < UNROLL_4; u++) {
    A_local[u] = A[tid + u * blockDim.x * gridDim.x];
  }
  for (int i = 0; i < iter; i++) {
    #pragma unroll
    for (u = 0; u < UNROLL_4; u++) {
      A_local[u]= fma(A_local[u], A_local[u], A_local[u]);
    }
   // A_local = A_local * A_local + A_local;
  }
  #pragma unroll
  for (u = 0; u < UNROLL_4; u++) {
    A[tid + u * blockDim.x * gridDim.x] = A_local[u];
  }
}

__global__ void execute_kernel_compute_cuda_kernel_unroll_8(long iter, double *A)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  double A_local[UNROLL_8];
  int u;
  #pragma unroll
  for (u = 0; u < UNROLL_8; u++) {
    A_local[u] = A[tid + u * blockDim.x * gridDim.x];
  }
  for (int i = 0; i < iter; i++) {
    #pragma unroll
    for (u = 0; u < UNROLL_8; u++) {
      A_local[u]= fma(A_local[u], A_local[u], A_local[u]);
    }
   // A_local = A_local * A_local + A_local;
  }
  #pragma unroll
  for (u = 0; u < UNROLL_8; u++) {
    A[tid + u * blockDim.x * gridDim.x] = A_local[u];
  }
}

__global__ void execute_kernel_compute_cuda_kernel_unroll_16(long iter, double *A)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  double A_local[UNROLL_16];
  int u;
  #pragma unroll
  for (u = 0; u < UNROLL_16; u++) {
    A_local[u] = A[tid + u * blockDim.x * gridDim.x];
  }
  for (int i = 0; i < iter; i++) {
    #pragma unroll
    for (u = 0; u < UNROLL_16; u++) {
      A_local[u]= fma(A_local[u], A_local[u], A_local[u]);
    }
   // A_local = A_local * A_local + A_local;
  }
  #pragma unroll
  for (u = 0; u < UNROLL_16; u++) {
    A[tid + u * blockDim.x * gridDim.x] = A_local[u];
  }
}
